#pragma once

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace tensorengine {

template<typename T>
__global__ void add(const T* A, const T* B, T* C, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        C[idx] = A[idx] + B[idx];
    }
}
}

#endif
