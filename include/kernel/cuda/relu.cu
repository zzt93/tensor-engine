#pragma once

#ifdef __HIPCC__
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

namespace tensorengine {

template<typename T>
__global__ void relu(const T* in, T* out, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        out[idx] = min(in[idx], T{0});
    }
}
}

#endif
