#ifdef __HIPCC__

#include "../../include/device.h"


namespace tensorengine {

    void* CUDADevice::allocate(size_t size) {
        void* ptr;
        CUDA_CHECK(hipMalloc(&ptr, size));
        return ptr;
    }

    void* CUDADevice::allocateAsync(size_t size, hipStream_t stream) {
        void* ptr;
        CUDA_CHECK(hipMallocAsync(&ptr, size, stream));
        return ptr;
    }

    void CUDADevice::free(void* ptr) {
        CUDA_CHECK(hipFree(ptr));
    }

    void CUDADevice::copy(void* dest, const void* src, size_t size) {
        CUDA_CHECK(hipMemcpy(dest, src, size, hipMemcpyDefault));
    }

    void CUDADevice::copyAsync(void* dest, const void* src, size_t size, hipStream_t stream) {
        CUDA_CHECK(hipMemcpyAsync(dest, src, size, hipMemcpyDefault, stream));
    }

    DeviceType CUDADevice::type() const {
        return DeviceType::CUDA;
    }

} // namespace tensorengine

#endif